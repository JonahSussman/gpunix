// Build with:
// nvcc driver_api.cu -arch=sm_61 -o driver_api.out -I/usr/local/cuda/include -L/usr/local/cuda/lib64 -lcuda

// Driver API
// https://stackoverflow.com/questions/22639097/what-is-the-difference-between-the-cuda-api-cu-and-cuda


#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <vector>


hipError_t err;

#define CHK(X) if ((err = X) != hipSuccess) printf("CUDA error %d at %d\n", (int)err, __LINE__)

int main(int argc, char *argv[]) {
  std::ifstream ptx_file("vectorAdd_kernel.ptx");
  if (!ptx_file) {
    std::cerr << "Error opening PTX file" << std::endl;
    return -1;
  }

  std::string ptx_string(
    (std::istreambuf_iterator<char>(ptx_file)), // Why surround with parens?
    std::istreambuf_iterator<char>()
  );

  std::cout << "PTX code:" << std::endl;
  std::cout << ptx_string << std::endl;


  hipCtx_t context;
  hipDevice_t cuDevice;

  // These next few lines simply initialize your work with the CUDA driver,
  // they're not specific to PTX compilation
  CHK(hipInit(0));
  CHK(hipDeviceGet(&cuDevice, 0)); // or some other device on your system
  CHK(hipCtxCreate(&context, 0, cuDevice));

  // The magic happens here:
  hipModule_t module;
  hipModuleLoadDataEx(&module, ptx_string.c_str(), 0, 0, 0);

  // And here is how you use your compiled PTX
  hipFunction_t VecAdd_kernel;
  hipModuleGetFunction(&VecAdd_kernel, module, "VecAdd_kernel");

  // Allocate/initialize vectors in host memory
  int N = 1024;
  size_t size = N * sizeof(float);

  std::vector<float> h_A(N, 1.0f);
  std::vector<float> h_B(N, 2.0f);
  std::vector<float> h_C(N);

  // Allocate vectors in device memory
  hipDeviceptr_t d_A, d_B, d_C;
  CHK(hipMalloc(&d_A, size));
  CHK(hipMalloc(&d_B, size));
  CHK(hipMalloc(&d_C, size));

  // Copy vectors from host memory to device memory
  CHK(hipMemcpyHtoD(d_A, h_A.data(), size));
  CHK(hipMemcpyHtoD(d_B, h_B.data(), size));

  const int threadsPerBlock = 256;
  const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  void* args[] = { &d_A, &d_B, &d_C, &N };

  CHK(hipModuleLaunchKernel(
    VecAdd_kernel,
    blocksPerGrid, 1, 1,
    threadsPerBlock, 1, 1,
    0,
    NULL,
    args, NULL
  ));

  // Copy result from device memory to host memory
  CHK(hipMemcpyDtoH(h_C.data(), d_C, size));

  // Verify result
  for (int i = 0; i < N; ++i) {
    if (h_C[i] != h_A[i] + h_B[i]) {
      std::cerr << "Error: C[" << i << "] = " << h_C[i] << std::endl;
      break;
    }
  }

  std::cout << "Result verified!" << std::endl;
}