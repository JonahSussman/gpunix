#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void test(
  volatile int *flag, 
  volatile int *data_ready, 
  volatile int *data
) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  while (true) {
    if (*flag == 0) {
      // wait for data transfer
      while (true) {
        if (*data_ready == 0) {
          printf("x");
        }
        else {
          break;
        }
      }
      printf("data %d\n", *data);
      __syncthreads();
    }
    else {
      break;
    }
  }

  printf("gpu finish %d\n", tid);
}

int main() {
  int attr = 0;
  hipDeviceGetAttribute(&attr, hipDeviceAttributeConcurrentManagedAccess, 0);
  if (attr == 0) {
    printf("GPU does not support cudaDevAttrConcurrentManagedAccess\n"); 
    return 0;
  }

  // flags
  int *flag;
  hipMallocManaged(&flag, sizeof(int));
  *flag = 0;

  int *data_ready;
  hipMallocManaged(&data_ready, sizeof(int));
  *data_ready = 0;

  // data
  int *data = (int*)malloc(sizeof(int));
  int *data_device;
  *data = 777;
  hipMalloc(&data_device, sizeof(int));
  hipMemcpy(data_device, data, sizeof(int), hipMemcpyHostToDevice);
  hipStream_t s1, s2;
  hipStreamCreate(&s1);
  hipStreamCreate(&s2);

  // launch kernel
  int block = 8, grid = 1;
  test<<<grid, block, 0, s1>>> (flag, data_ready, data_device);

  // random host code
  for (int i = 0; i < 1e5; i++);
  printf("host do something\n");

  // update data
  *data = 987;
  hipMemcpyAsync(data_device, data, sizeof(int), hipMemcpyHostToDevice, s2);
  printf("host copied\n");
  *data_ready = 1;

  // update flag
  *flag = 1;

  hipDeviceSynchronize();

  // free memory
  hipFree(flag);

  printf("host finish\n");
}