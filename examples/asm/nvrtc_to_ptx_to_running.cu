#include <string>
#include <iostream>
#include <stdexcept>

#include <hip/hip_runtime.h>

#include <hip/hiprtc.h>

hipError_t cuda_err;
hiprtcResult nvrtc_err;

#define CHK_CUDA(X) \
  if ((cuda_err = X) != hipSuccess) \
    printf("CUDA error %d at %d\n", (int)cuda_err, __LINE__)

#define CHK_NVRTC(X) \
  if ((nvrtc_err = X) != HIPRTC_SUCCESS) \
    printf("NVRTC error %d at %d\n", (int)nvrtc_err, __LINE__)

std::string compile_to_ptx(const std::string& source_code) {
    // Compile the source code to PTX using nvrtc
    hiprtcProgram prog;
    CHK_NVRTC(hiprtcCreateProgram(
      &prog, 
      source_code.c_str(), 
      "kernel.cu", 
      0, 
      nullptr, 
      nullptr
    ));
    
    hiprtcResult compile_result = hiprtcCompileProgram(prog, 0, nullptr);
    if (compile_result != HIPRTC_SUCCESS) {
        size_t log_size;
        hiprtcGetProgramLogSize(prog, &log_size);
        std::string log(log_size, '\0');
        hiprtcGetProgramLog(prog, &log[0]);
        throw std::runtime_error("Compilation failed: " + log);
    }
    
    size_t ptx_size;
    hiprtcGetCodeSize(prog, &ptx_size);
    std::string ptx(ptx_size, '\0');
    hiprtcGetCode(prog, &ptx[0]);
    
    hiprtcDestroyProgram(&prog);
    
    return ptx;
}